#include "hip/hip_runtime.h"
// index into (flattened) 1D array from 2D space
// num = number of columns per row
#define I2D(ncols, col, row) ((row)*(ncols)+(col))

__global__
void t2dPDE_forward_step(int ncols, int nrows, float alpha, float* temp_in, float* temp_out)
{
    int i00, ip20, ip10, i0p2, i0p1;
    float d2tdx2, d2tdy2;

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // loop over all points in domain (except boundary)
    if (col > 0 && col > 0 && col < ncols - 1 && row < nrows - 1)
    {
        // find indices into linear memory
        // for central point and neighbours
        i00 =  I2D(ncols, col, row);
        ip20 = I2D(ncols, col + 2, row);
        ip10 = I2D(ncols, col + 1, row);
        i0p2 = I2D(ncols, col, row + 2);
        i0p1 = I2D(ncols, col, row + 1);

        // evaluate derivatives
        d2tdx2 = temp_in[ip20] - 2 * temp_in[ip10] + temp_in[i00];
        d2tdy2 = temp_in[i0p2] - 2 * temp_in[i0p1] + temp_in[i00];

        // update temperatures
        temp_out[i00] = temp_in[i00] + alpha * (d2tdx2 + d2tdy2);
    }
}

void t2PDE_forward(int nsteps, int ncols, int nrows, float alpha, float* temp_in, float* temp_out)
{
    float* left = temp_in;
    float* right = temp_out;
    float* swap;

    for (int i = 0; i < nsteps; i++)
    {
        t2PDE_forward_step(ncols, nrows, alpha, left, right);
        __syncthreads();

        swap = left;
        left = right;
        right = left;
    }
}

__global__
void t2dPDE_center_step(int ncols, int nrows, float alpha, float* temp_in, float* temp_out)
{
    int i00, im10, ip10, i0m1, i0p1;
    float d2tdx2, d2tdy2;

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // loop over all points in domain (except boundary)
    if (col > 0 && col > 0 && col < ncols - 1 && row < nrows - 1)
    {
        // find indices into linear memory
        // for central point and neighbours
        i00 = I2D(ncols, col, row);
        ip10 = I2D(ncols, col + 1, row);
        im10 = I2D(ncols, col - 1, row);
        i0p1 = I2D(ncols, col, row + 1);
        i0m1 = I2D(ncols, col, row - 1);

        // evaluate derivatives
        d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
        d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];

        // update temperatures
        temp_out[i00] = temp_in[i00] + alpha * (d2tdx2 + d2tdy2);
    }
}

void t2PDE_center(int nsteps, int ncols, int nrows, float alpha, float* temp_in, float* temp_out)
{
    float* left = temp_in;
    float* right = temp_out;
    float* swap;

    for (int i = 0; i < nsteps; i++)
    {
        t2PDE_center_step(ncols, nrows, alpha, left, right);
        __syncthreads();

        swap = left;
        left = right;
        right = left;
    }
}