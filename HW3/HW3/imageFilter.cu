#include "hip/hip_runtime.h"
extern "C" __global__ void imageFilterSingle(
    float* d_output, float* d_input, float* d_filter,
    int num_row, int num_col, int filter_size)
{
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx_x < num_col && idx_y < num_row)
    {
        float result = 0.f;
        for (int filter_row = -filter_size / 2; filter_row <= filter_size / 2; ++filter_row)
        {
            for (int filter_col = -filter_size / 2; filter_col <= filter_size / 2; ++filter_col)
            {
                int image_row = idx_y + filter_row;
                int image_col = idx_x + filter_col;
                float filter_value = d_filter[(filter_row + filter_size / 2) * filter_size + filter_col + filter_size / 2];
                float image_value;

                if (image_row >= 0 && image_row < num_row && image_col >= 0 && image_col < num_col)
                {
                    image_value = d_input[image_row * num_col + image_col];
                }
                else
                {
                    image_value = 0.f;
                }

                result += image_value * filter_value;
            }

        }

        d_output[idx_y * num_col + idx_x] = result;
    }
}

extern "C" __global__ void imageFilter(
    float* d_output, float* d_input,
    int num_row, int num_col,
    float* filters, int* filter_sizes, int nfilters)
{
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx_x < num_col && idx_y < num_row)
    {
        float* cursor = filters;

        for (int i = 0; i < nfilters; i++)
        {
            int filterSize = filter_sizes[i];

            imageFilterSingle(d_output, d_input, cursor, num_row, num_col, filterSize);

            cursor += filterSize * filterSize;

            __syncthreads();
        }
    }
}
