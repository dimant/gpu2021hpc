
#include <hip/hip_runtime.h>
extern "C" __global__ void blur3x3(unsigned char* A, unsigned char* B, int rows, int cols)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col < cols && row < rows)
	{
		int pixVal = 0;
		int pixels = 0;

		for (int blurRow = -1; blurRow < 1 + 1; blurRow++)
		{
			for (int blurCol = -1; blurCol < 1 + 1; blurCol++)
			{
				int curRow = row + blurRow;
				int curCol = col + blurCol;

				if (curRow > -1 && curRow < rows && curCol > -1 && curCol < cols)
				{
					pixVal += A[curRow * cols + curCol];
					pixels++;
				}
			}
		}

		B[row * cols + col] = (unsigned char)(pixVal / pixels);
	}
}

extern "C" __global__ void blur9x9(unsigned char* A, unsigned char* B, int rows, int cols)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col < cols && row < rows)
	{
		int pixVal = 0;
		int pixels = 0;

		for (int blurRow = -4; blurRow < 4 + 1; blurRow++)
		{
			for (int blurCol = -4; blurCol < 4 + 1; blurCol++)
			{
				int curRow = row + blurRow;
				int curCol = col + blurCol;

				if (curRow > -1 && curRow < rows && curCol > -1 && curCol < cols)
				{
					pixVal += A[curRow * cols + curCol];
					pixels++;
				}
			}
		}

		B[row * cols + col] = (unsigned char)(pixVal / pixels);
	}
}
