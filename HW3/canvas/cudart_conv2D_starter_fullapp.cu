#include "hip/hip_runtime.h"
﻿
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

/**
 * CUDA Kernels
 *
 */
 // 2D Convolution kernel "naive" global memory
__global__ void conv_2D_allglobal(float* d_output, float* d_input, float* d_filter, int num_row, int num_col, int filter_size)
{
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx_x < num_col && idx_y < num_row)
    {
        float result = 0.f;
        for (int filter_row = -filter_size / 2; filter_row <= filter_size / 2; ++filter_row)
        {
            for (int filter_col = -filter_size / 2; filter_col <= filter_size / 2; ++filter_col)
            {
                int image_row = idx_y + filter_row;
                int image_col = idx_x + filter_col;
                float image_value = (image_row >= 0 && image_row < num_row && image_col >= 0 && image_col < num_col) ? d_input[image_row * num_col + image_col] : 0.f;
                float filter_value = d_filter[(filter_row + filter_size / 2) * filter_size + filter_col + filter_size / 2];
                result += image_value * filter_value;
            }
        }
        d_output[idx_y * num_col + idx_x] = result;
    }

}


using namespace std;


float gaussBlurFilter_7x7[49] = {
   0.0086f / 3.0f,    0.0198f / 3.0f,    0.0326f / 3.0f,    0.0386f / 3.0f,    0.0326f / 3.0f,    0.0198f / 3.0f,    0.0086f / 3.0f,
   0.0198f / 3.0f,    0.0456f / 3.0f,    0.0751f / 3.0f,    0.0887f / 3.0f,    0.0751f / 3.0f,    0.0456f / 3.0f,    0.0198f / 3.0f,
   0.0326f / 3.0f,    0.0751f / 3.0f,	  0.1239f / 3.0f,    0.1463f / 3.0f,    0.1239f / 3.0f,    0.0751f / 3.0f,    0.0326f / 3.0f,
   0.0386f / 3.0f,    0.0887f / 3.0f,    0.1463f / 3.0f,    0.1729f / 3.0f,    0.1463f / 3.0f,    0.0887f / 3.0f,    0.0386f / 3.0f,
   0.0326f / 3.0f,    0.0751f / 3.0f,    0.1239f / 3.0f,	   0.1463f / 3.0f,    0.1239f / 3.0f,    0.0751f / 3.0f,    0.0326f / 3.0f,
   0.0198f / 3.0f,    0.0456f / 3.0f,    0.0751f / 3.0f,    0.0887f / 3.0f,    0.0751f / 3.0f,    0.0456f / 3.0f,    0.0198f / 3.0f,
   0.0086f / 3.0f,    0.0198f / 3.0f,    0.0326f / 3.0f,    0.0386f / 3.0f,    0.0326f / 3.0f,    0.0198f / 3.0f,    0.0086f / 3.0f
};

float gaussBlurFilter_5x5[25] = {
    1.0f / 273.0f, 4.0f / 273.0f, 7.0f / 273.0f, 4.0f / 273.0f, 1.0f / 273.0f,
    4.0f / 273.0f, 16.0f / 273.0f, 26.0f / 273.0f, 16.0f / 273.0f, 4.0f / 273.0f,
    7.0f / 273.0f, 26.0f / 273.0f, 41.0f / 273.0f, 26.0f / 273.0f, 7.0f / 273.0f,
    4.0f / 273.0f, 16.0f / 273.0f, 26.0f / 273.0f, 16.0f / 273.0f, 4.0f / 273.0f,
    1.0f / 273.0f, 4.0f / 273.0f, 7.0f / 273.0f, 4.0f / 273.0f, 1.0f / 273.0f
};

float compositeLaplacianFilter[9] = {
    -1.0f, -1.0f, -1.0f,
    -1.0f, 9.0f, -1.0f,
    -1.0f, -1.0f, -1.0f
};

float basicLaplacianFilterDiags[9] = {
    1.0f, 1.0f, 1.0f,
    1.0f, -8.0f, 1.0f,
    1.0f, 1.0f, 1.0f
};

float sobelEdgeX[9] = {
    1.0f, 0.0f, -1.0f,
    2.0f, 0.0f, -2.0f,
    1.0f, 0.0f, -1.0f
};


int filterWidth = 7;


/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int imgRows;
    int imgCols;
    int imgChannels;

    string strOutpath = "c:\\users\\photonuser\\Downloads\\curt_2Dconv_kernels\\";

    string inFile = "Big-Gray-Lena_8bit.png";
    
    unsigned char* h_inimg_char = stbi_load( (strOutpath + inFile).c_str(), &imgCols, &imgRows, &imgChannels, 0);
    //unsigned short *data16 = stbi_load_16("C:/Users/colinrein/Pictures/boat1_16bit.png", &imgRows, &imgCols, &imgChannels, 0);
    // REMEMBER: stbi_image_free(imgdata);

    int numElements = imgCols * imgRows;
    size_t size = numElements * sizeof(float);
    printf("[Convolution Filter of [%d x %d] image with filter]\n", imgRows, imgCols);

    float* h_inimg = (float*)malloc(size);
    for (int i = 0; i < numElements; i++)
    {
        h_inimg[i] = (int)h_inimg_char[i]; // convert from char to float, costly...
    }

    float* h_outimg = (float*)malloc(size);
    unsigned char* h_outimg_char = (unsigned char*)malloc(imgRows * imgCols * imgChannels * sizeof(unsigned char));
 
    // Verify that allocations succeeded
    if (h_inimg == NULL || h_outimg == NULL )
    {
        fprintf(stderr, "Failed to allocate host images!\n");
        exit(EXIT_FAILURE);
    }

	// Allocate memory for device input image, filter, output image
	// and make sure it is initialized with the correct data

    // Launch the CUDA Kernel
    dim3 threadsPerBlock{ 16,16,1 };
    dim3 blocksPerGrid{ (imgCols + threadsPerBlock.x - 1) / threadsPerBlock.x, (imgRows + threadsPerBlock.y - 1) / threadsPerBlock.y, 1 };
    printf("CUDA kernel launch exec cfg with (%d,%d,%d) blocks of (%d,%d,%d) threads\n", blocksPerGrid.x, blocksPerGrid.y, blocksPerGrid.z, threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);
 
    conv_2D_allglobal <<< blocksPerGrid, threadsPerBlock >>> (d_outimg, d_inimg, d_filt, imgRows, imgCols, filterWidth);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

	// retrieve result image from device
 

    // convert float to char (lossy...)
    for (int i = 0; i < numElements; i++)
    {
        h_outimg_char[i] = (unsigned char)(unsigned int)h_outimg[i]; // convert from float to 8-bit char, costly...
    }


    string strOutfile = "result_image.png";
    int stbErr = stbi_write_jpg((strOutpath + strOutfile).c_str(), imgCols, imgRows, imgChannels, h_outimg_char, 100);

    // Free device global memory


    // Free host memory
    stbi_image_free(h_inimg);
    free(h_outimg);

    printf("Done\n");
    return 0;
}
