
#include <hip/hip_runtime.h>
extern "C" __global__ void transpose(int* A, int* B, int rows, int cols)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int index = row * rows + col;
	int transposedIndex = col * rows + row;

	if (col < cols && row < rows)
	{
		B[index] = A[transposedIndex];
	}
}