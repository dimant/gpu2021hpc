#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include "nn_cuda_kernels.cuh"

#include <stdio.h>
#include <cassert>
#include <random>

inline hipError_t CURT_CHK(hipError_t curtRes)
{
    if (hipSuccess != curtRes)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(curtRes));
        assert(curtRes == hipSuccess);
    }
    return curtRes;
}

#include "mnist_dataset_helper.h"
#include "nn_host_helper.h"

#define LOAD_NETWORK_PARAMETERS  0


// forward-declarations
void run_fullyconnected_neuralnet(bool bLoadParams, bool bSaveParams, float* mnist_input, float* mnist_targets, int num_samples);
void run_convolutional_neuralnet(bool bLoadParams, bool bSaveParams, float* mnist_input, float* mnist_targets, int num_samples);

//===================================================================================================
//============== Main Host Application ================
int main()
{
    // =========== LOAD MNIST datasets ================================
    // Assume MNIST dataset is in C:\FinalProject\mnist sub-folders
    // TODO: update full-paths if necessary
    string mnist_test_imgs_filepath("C:\\FinalProject\\mnist\\t10k-images-idx3-ubyte\\t10k-images.idx3-ubyte");
    string mnist_test_labels_filepath("C:\\FinalProject\\mnist\\t10k-labels-idx1-ubyte\\t10k-labels.idx1-ubyte");
    string mnist_train_imgs_filepath("C:\\FinalProject\\mnist\\train-images-idx3-ubyte\\train-images.idx3-ubyte");
    string mnist_train_labels_filepath("C:\\FinalProject\\mnist\\train-labels-idx1-ubyte\\train-labels.idx1-ubyte");

    // Functions will malloc memory for arrays, host/caller must free() it.
    // these int values will be filled with what is read from the MNIST dataset files
    int num_test_imgs = 0;
    int num_train_imgs = 0;
    int num_test_lbls = 0;
    int num_train_lbls = 0;
    int n_rows = 0;
    int n_cols = 0;
    //// loads data into one contiguous 1D packed memory region, sample-by-sample, row-major layout
    // each sample label data is a one-hot-encoded vector of length = 10 where the "correct" digit = 1 and all others = 0
    float* X0_test = NULL;  // test images = input
    float* T_test = NULL;  // test labels (output targets)
    float* X0_train = NULL; // train images = input
    float* T_train = NULL; // train labels (output targets)

    load_preproc_mnist_labels(mnist_test_labels_filepath, &T_test, num_test_lbls); // test set - targets
    load_preproc_mnist_labels(mnist_train_labels_filepath, &T_train, num_train_lbls); // train set - targets
    load_and_preproc_mnist_images(mnist_test_imgs_filepath, &X0_test, num_test_imgs, n_rows, n_cols); // test set - images
    load_and_preproc_mnist_images(mnist_train_imgs_filepath, &X0_train, num_train_imgs, n_rows, n_cols); // train set - images
    //===============================================
 
    // 
    // TODO: Choose which GPU to run on, make sure this agrees with your system (works for AWS VDE).
    CURT_CHK(hipSetDevice(0));

    //===== TRAINING
    run_fullyconnected_neuralnet(false,true, X0_train, T_train, num_train_imgs);
    //run_convolutional_neuralnet(false, false, X0_train, T_train);

    //===== TESTING (after training is done and final network parameters are saved)
    //run_fullyconnected_neuralnet(false, false, X0_test, T_test);
    //run_convolutional_neuralnet(false, false, X0_test, T_test);

    //===============================================

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    CURT_CHK(hipDeviceReset());

    return 0;

};


void run_fullyconnected_neuralnet(bool bLoadParams, bool bSaveParams, float* mnist_input, float* mnist_targets, int num_samples)
{
    // A simple fully-connected NN architecture
    // INPUT layer (0) : 28x28 MNIST image
    // FC layer 1 : 128 nodes, sigmoid or relu activation :             100,352 weights, 128 bias
    // FC layer 2 : 128 nodes, sigmoid or relu  activation               16,512 weights, 128 bias
    // Output FC layer (3) : 10 nodes, softmax activation                  1280 weights, 10 bias
    // Expected test accuracy ~ 99%

    const int NUM_NN_PARAM_ARRAYS = 6;

    string strNNparamfile("C:\\FinalProject\\my_FC_NN_params.bin");  // USE a unique name which you can remember which network arch it's for!

    // Define layer sizes array with the size (in # floats) for each parameter: weights and biases get separate entries.
    int layer_sizes[] = { 100352, 128, 16384, 128, 1280, 10 };  // W1, b1, W2, b2, W3, b3
    
    // allocate memory for the array of layerParamArray structures - which define the size and base address of each weight/bias array
    layerParamArray* pParamArrayLoad = (layerParamArray*)malloc(NUM_NN_PARAM_ARRAYS * sizeof(layerParamArray));
    for (int i = 0; i < NUM_NN_PARAM_ARRAYS; ++i)
    {
        pParamArrayLoad[i].num_floats = layer_sizes[i];
        // NOTE: these arrays get passed to GPU so allocate with CUDA memory (unified managed memory)
        CURT_CHK(hipMallocManaged(&(pParamArrayLoad[i].pArray), layer_sizes[i] * sizeof(float)));
        // also allocate array of same size for error gradient used during backprop (will not be persisted with network parameters)
        CURT_CHK(hipMallocManaged(&(pParamArrayLoad[i].pGradArray), layer_sizes[i] * sizeof(float)));
    }

    if (bLoadParams) // LOAD existing save network parameters
    {
        //=== LOADING network parameters from a binary file

        if (0 != load_network_parameters_from_binaryfile(strNNparamfile, NUM_NN_PARAM_ARRAYS, pParamArrayLoad))
        {
            fprintf(stderr, "Error loading network parameters from file: %s.\n", strNNparamfile.c_str());
        }
    }
    else  // INIT new params
    {
       default_random_engine generator;
       for (int i = 0; i < NUM_NN_PARAM_ARRAYS; ++i)
        {
           float stddev = 0;
           if (0 == i)
               stddev = sqrtf(2.0f / 784); // He initializer
           else
               stddev = sqrt(2.0f / layer_sizes[i - 1]);

           if (0 == (i % 2)) // weight arrays are even indices
           {
               normal_distribution<float> normD(0.0f, stddev);
               for (int j = 0; j < layer_sizes[i]; ++j)
               {
                   pParamArrayLoad[i].pArray[j] = normD(generator);
               }
           }
           else // bias - init to zeros
               CURT_CHK(hipMemset(pParamArrayLoad[i].pArray, 0, layer_sizes[i] * sizeof(float)));
        }

    }

    // Set up arrays for intermediate network data values (used for backprop)
    float* Z1 = NULL, float* Z2 = NULL, float* Z3 = NULL;
    float* Y1 = NULL, float* Y2 = NULL, float* Y3 = NULL;
    float* dOut = NULL, float* dY3 = NULL, float* dY2 = NULL, float* dY1 = NULL;

    CURT_CHK(hipMallocManaged(&Z1, 128 * sizeof(float)));
    CURT_CHK(hipMallocManaged(&Z2, 128 * sizeof(float)));
    CURT_CHK(hipMallocManaged(&Z3, 10 * sizeof(float)));
    CURT_CHK(hipMallocManaged(&Y1, 128 * sizeof(float)));
    CURT_CHK(hipMallocManaged(&Y2, 128 * sizeof(float)));
    CURT_CHK(hipMallocManaged(&Y3, 10 * sizeof(float)));
    CURT_CHK(hipMallocManaged(&dY3, 10 * sizeof(float)));
    CURT_CHK(hipMallocManaged(&dY2, 128 * sizeof(float)));
    CURT_CHK(hipMallocManaged(&dY1, 128 * sizeof(float)));

    // TODO: MAIN FC NN Implementation here
    for(int s = 0; s < num_samples; ++s)
    {
        // Forward pass
        // TODO: define correct kernel execution configs for each layer, based on layer cuda kernel implementations
        // TODO: add additional input parameters to kernels if necessary
        dim3 numBlocks;
        dim3 numThreadsPerBlock;
        nn_layer_fullyconnected_forward << <numBlocks, numThreadsPerBlock >> > (mnist_input, pParamArrayLoad[0].pArray, pParamArrayLoad[1].pArray, Z1, Y1);
        nn_layer_fullyconnected_forward << <numBlocks, numThreadsPerBlock >> > (Y1, pParamArrayLoad[2].pArray, pParamArrayLoad[3].pArray, Z2, Y2);
        nn_layer_fullyconnected_forward << <numBlocks, numThreadsPerBlock >> > (Y2, pParamArrayLoad[4].pArray, pParamArrayLoad[5].pArray, Z3, Y3);

        //// Backward pass
        // TODO: define correct kernel execution configs for each layer, based on layer cuda kernel implementations
        // TODO: add additional input parameters to kernels if necessary
        nn_layer_output_layer_backward << <numBlocks, numThreadsPerBlock >> > (mnist_targets, Y3, Z3, dY3);
        nn_layer_fullyconnected_backward << <numBlocks, numThreadsPerBlock >> > (dY3, pParamArrayLoad[4].pArray, Y2, Z2, dY2, pParamArrayLoad[4].pGradArray, pParamArrayLoad[5].pGradArray  );
        nn_layer_fullyconnected_backward << <numBlocks, numThreadsPerBlock >> > (dY2, pParamArrayLoad[2].pArray, Y1, Z1, dY1, pParamArrayLoad[2].pGradArray, pParamArrayLoad[3].pGradArray);
        nn_layer_fullyconnected_backward << <numBlocks, numThreadsPerBlock >> > (dY1, pParamArrayLoad[0].pArray, mnist_input, NULL, NULL, pParamArrayLoad[0].pGradArray, pParamArrayLoad[1].pGradArray);

        // TODO: put numerical gradient checking here - be sure to disable for real complete training & test runs

    }

    if (bSaveParams)
    {
        //=== SAVING all network parameters to a binary file
         
        if (0 != save_network_parameters_to_binaryfile(strNNparamfile, NUM_NN_PARAM_ARRAYS, pParamArrayLoad))
        {
            fprintf(stderr, "Error saving network parameters to file: %s.\n", strNNparamfile.c_str());
        }
    }

};

void run_convolutional_neuralnet(bool bLoadParams, bool bSaveParams, float* mnist_input, float* mnist_targets, int num_samples)
{
    // A simple convolutional NN architecture
    // INPUT layer (0): 28x28 MNIST image
    // CONV layer 1 : 32 output FM, 3x3 filter, relu activation:        288 weights, 32 bias
    // MAX POOL layer 2 : 2x2 pooling                                   0
    // CONV layer 3 : 64 output FM, 3x3 filter, relu activation         18432 weights, 64 bias
    // MAX POOL layer 4 : 2x2 pooling                                   0
     // OUTPUT FC layer (5) : 10 nodes, softmax activation:             16000 weights, 10 bias  
    // Expected test accuracy ~ %99

    float* W1 = NULL, float* W2 = NULL, float* W3 = NULL, float* b1 = NULL, float* b2 = NULL, float* b3 = NULL;

    string strNNparamfile("C:\\FinalProject\\my_CONV_NN_params.bin");  // USE a unique name which you can remember which network arch it's for!
    const int NUM_NN_PARAM_ARRAYS = 6;
    // Set up layer sizes array with the size (in # floats) for each parameter: weights and biases get separate entries.
    int layer_sizes[] = { 288, 32, 18432, 64, 16000, 10 };

    // allocate memory for the array of layerParamArray structures - which define the size and base address of each weight/bias array
    layerParamArray* pParamArrayLoad = (layerParamArray*)malloc(NUM_NN_PARAM_ARRAYS * sizeof(layerParamArray));
    for (int i = 0; i < NUM_NN_PARAM_ARRAYS; ++i)
    {
        pParamArrayLoad[i].num_floats = layer_sizes[i];
        // NOTE: these arrays get passed to GPU so allocate with CUDA memory (unified managed memory)
        CURT_CHK(hipMallocManaged(&(pParamArrayLoad[i].pArray), layer_sizes[i] * sizeof(float)));
    }

    if (bLoadParams)
    {
        //=== LOADING network parameters from a binary file
        if (0 != load_network_parameters_from_binaryfile(strNNparamfile, NUM_NN_PARAM_ARRAYS, pParamArrayLoad))
        {
            fprintf(stderr, "Error loading network parameters from file: %s.\n", strNNparamfile.c_str());
        }
    }
    else  // INIT new params
    {

    }

    // TODO: MAIN CONV NN Implementation here

    if (bSaveParams)
    {
        //=== SAVING network parameters to a binary file
        // pass the pointer to base address of each parameter array (pParams is an array of pointers)
        if (0 != save_network_parameters_to_binaryfile(strNNparamfile, NUM_NN_PARAM_ARRAYS, pParamArrayLoad))
        {
            fprintf(stderr, "Error saving network parameters to file: %s.\n", strNNparamfile.c_str());
        }
    }

};

