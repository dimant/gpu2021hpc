#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <>

#include "fc_forward.h"

__device__ __host__ __forceinline__ float relu(const float f)
{
    return (fmaxf(0, f));
}

__device__ __host__ __forceinline__ float relu_prime(const float f)
{
    if (0.f == f)
        return 0.5f;
    else
        return (f < 0.f ? 0.0f : 1.0f);
}

__device__ __host__ __forceinline__ float softmax10(const float* fvec10, const int ti)
{
    float sum = 0;

    float target = expf(fvec10[ti]);

#pragma unroll
    for (int i = 0; i < 10; i++)
    {
        sum += expf(fvec10[i]);
    }
    return (target / sum);
}

__device__ __host__ __forceinline__ float softmax10_prime(const float* fvec10, const int ds_i)
{
    float si = softmax10(fvec10, ds_i);
    return si * (1 - si);
}

__global__ void fc_forward(const float* X, const float* W, const float* b, float* Z, float* Y, const int inputs, const int activation)
{
    int neuron = threadIdx.x;

    float sum = b[neuron];

    for (int input = 0; input < inputs; input++)
    {
        sum += X[input] * W[neuron * inputs + input];
    }

    Z[neuron] = sum;

    switch (activation)
    {
        case ACT_RELU:
            Y[neuron] = relu(sum);
            break;
        case ACT_SOFTMAX10:
            __syncthreads();
            Y[neuron] = softmax10(Z, neuron);
            break;
    }
}

// last layer: deltaL = (y hat - y) activation function'(previous layer activation)
__global__ void output_backward(
    const float* labels,
    const float* Yhat,
    const float* Z,
    float* dY)
{
    int neuron = threadIdx.x;

    dY[neuron] = (labels[neuron] - Yhat[neuron]) * softmax10_prime(Z, neuron);
}

// any other layer: 
// https://machinelearningmastery.com/implement-backpropagation-algorithm-scratch-python/
// https://cloudacademy.com/course/data-machine-learning-gradient-descent/derivative-calculation/
// https://medium.com/@14prakash/back-propagation-is-very-simple-who-made-it-complicated-97b794c97e5c
__global__ void fc_backward(
    const float* dY, // errors for current layer
    const float* Yprev,
    const float* Zprev,
    const float* W,
    float* dW,
    float* dB,
    float* dX, // errors for next layer
    const int inputs,
    const int neurons)
{
    int neuron = threadIdx.x;

    for (int input = 0; input < inputs; input++)
    {
        dW[neuron * inputs + input] = dY[neuron] * Yprev[input];
        dX[input] += W[neuron * inputs + input] * dY[input];
    }

    dB[neuron] = dY[neuron];

    __syncthreads();

    if (neuron == 0)
    {
        for (int input = 0; input < inputs; input++)
        {
            dX[input] *= relu_prime(Zprev[input]);
        }
    }
}
