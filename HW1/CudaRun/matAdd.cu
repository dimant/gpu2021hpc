
#include <hip/hip_runtime.h>


__global__ void matAdd(const float* A, const float* B, float* C, int nrows, int ncols)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < nrows && col < ncols)
    {
        int idx = row * ncols + col;
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void matAddRow(const float* A, const float* B, float* C, int nrows, int ncols)
{
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < nrows)
    {
        for (int col = 0; col < ncols; col++)
        {
            int idx = row * ncols + col;
            C[idx] = A[idx] + B[idx];
        }
    }
}

__global__ void matAddCol(const float* A, const float* B, float* C, int nrows, int ncols)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (col < ncols)
    {
        for (int row = 0; row < nrows; row++)
        {
            int idx = row * ncols + col;
            C[idx] = A[idx] + B[idx];
        }
    }
}

