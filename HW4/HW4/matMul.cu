
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

__global__ void matMulGlobal(float* A, float* B, float* C, int width)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < width && col < width)
	{
		float matmul = 0;

		for (int i = 0; i < width; ++i)
		{
			matmul += A[row * width + i] * B[i * width + col];
		}

		C[row * width + col] = matmul;
	}
}

__global__ void matMulShared(float* A, float* B, float* C, int width)
{
	__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int row = blockIdx.y * blockDim.y + ty;
	int col = blockIdx.x * blockDim.x + tx;
	
	float matmul = 0;

	// iterates over matrix in strides of size TILE_WIDTH
	// the iteration happens in x and y direction because the indices are dependent on tx and ty
	for (int p = 0; p < width / TILE_WIDTH; ++p)
	{
		// load all cells of the tile. each cell is loaded by a different thread
		ds_A[ty][tx] = A[row * width + p * TILE_WIDTH + tx];
		ds_B[ty][tx] = B[(p * TILE_WIDTH + ty) * width + col];

		__syncthreads();

		// calculate partial sum for tile.
		for (int i = 0; i < TILE_WIDTH; ++i)
		{
			matmul += ds_A[ty][i] * ds_B[i][tx];
		}

		__syncthreads();
	}

	C[row * width + col] = matmul;
}

