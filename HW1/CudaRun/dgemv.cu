
#include <hip/hip_runtime.h>
extern "C" __global__ void dgemv(
    const double alpha,
    const double* A,
    const double* x,
    const double beta,
    const double* y,
    double* z,
    size_t rows, size_t cols)
{
    int stride = blockDim.x * gridDim.x;
    double buf;

    for (int row = blockDim.x * blockIdx.x + threadIdx.x; row < rows; row += stride)
    {
        buf = 0.0;

        for (int col = 0; col < cols; col++)
        {
            buf += alpha * A[row * cols + col] * x[col];
        }

        z[row] = buf + beta * y[row];
    }
}