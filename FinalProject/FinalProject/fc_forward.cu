#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include <>

#include "fc_forward.h"

__device__ __host__ __forceinline__ float relu(float f)
{
    return (fmaxf(0, f));
}

__device__ __host__ __forceinline__ float reluPrime(float f)
{
    if (0.f == f)
        return 0.5f;
    else
        return (f < 0.f ? 0.0f : 1.0f);
}

__device__ __host__ __forceinline__ float softmax10(float* fvec10, int ti)
{
    float sum = 0;

    float target = expf(fvec10[ti]);

#pragma unroll
    for (int i = 0; i < 10; i++)
    {
        sum += expf(fvec10[i]);
    }
    return (target / sum);
}

// fvec10: activations from previous layer
// ds_i: perceptron index within the layer
// ds_k: index of vector element towards which we are taking a partial derivative
__device__ __host__ __forceinline__ float softmax10Prime(float* fvec10, int ds_i)
{
    float si = softmax10(fvec10, ds_i);
    return si * (1 - si);
}


__global__ void fc_forward(const float* X, const float* W, const float* b, float* Z, float* Y, const int n, const int activation)
{
    int wi = threadIdx.x;

    float sum = b[wi];

    for (int xi = 0; xi < n; xi++)
    {
        sum += X[xi] * W[wi * n + xi];
    }

    Z[wi] = sum;

    switch (activation)
    {
        case ACT_RELU:
            Y[wi] = relu(sum);
            break;
        case ACT_SOFTMAX10:
            __syncthreads();
            Y[wi] = softmax10(Z, wi);
            break;
    }
}

__global__ void output_backward(float* labels, float* Yhat, float* Z, int n, float* dY)
{
    int ds_i = threadIdx.x;

    dY[ds_i] = (labels[ds_i] - Yhat[ds_i]) * softmax10Prime(Z, ds_i);
}

// last layer: deltaL = (y hat - y) activation function'(previous layer activation)
// any other layer: 
// https://machinelearningmastery.com/implement-backpropagation-algorithm-scratch-python/
// https://cloudacademy.com/course/data-machine-learning-gradient-descent/derivative-calculation/
// https://medium.com/@14prakash/back-propagation-is-very-simple-who-made-it-complicated-97b794c97e5c