#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void childKernel_level2() 
{
	printf("Dynamic "); 
} 

__global__ void childKernel_level1() 
{
	printf("CUDA ");
	childKernel_level2<<< 1, 1 >>> ();
	// get errors and wait for child to complete 
	if (hipSuccess != hipGetLastError()) { return; }
	if (hipSuccess != hipDeviceSynchronize()) { return; }
}
__global__ void parentKernel() 
{
printf("Hello ");
// launch child 
childKernel_level1<<<1, 1 >>> ();
// get errors and wait for child to complete 
if (hipSuccess != hipGetLastError()) { return; }
if (hipSuccess != hipDeviceSynchronize()) { return; }
printf("Parallelism!\n"); 
} 

int main(int argc, char* argv[]) 
{
	// launch parent 
	parentKernel<<<1, 1 >>> ();
	if (hipSuccess != hipGetLastError()) { return 1; }
	// wait for parent to complete 
	if (hipSuccess != hipDeviceSynchronize())
	{
		return 2;
	}
	return 0;
}