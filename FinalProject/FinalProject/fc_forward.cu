#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include <>

#include "fc_forward.h"

__device__ __host__ __forceinline__ float relu(float f)
{
    return (fmaxf(0, f));
}

__device__ __host__ __forceinline__ float softmax10(float* fvec10, int ti)
{
    float sum = 0;

    float target = expf(fvec10[ti]);

#pragma unroll
    for (int i = 0; i < 10; i++)
    {
        sum += expf(fvec10[i]);
    }
    return (target / sum);
}

__global__ void fc_forward(const float* X, const float* W, const float* b, float* Z, float* Y, const int n, const int activation)
{
    int wi = threadIdx.x;

    float sum = b[wi];

    for (int xi = 0; xi < n; xi++)
    {
        sum += X[xi] * W[wi * n + xi];
    }

    Z[wi] = sum;

    switch (activation)
    {
        case ACT_RELU:
            Y[wi] = relu(sum);
            break;
        case ACT_SOFTMAX10:
            __syncthreads();
            Y[wi] = softmax10(Z, wi);
            break;
    }
}

// last layer: deltaL = (y hat - y) activation function'(previous layer activation)
// any other layer: 
// https://machinelearningmastery.com/implement-backpropagation-algorithm-scratch-python/
// https://cloudacademy.com/course/data-machine-learning-gradient-descent/derivative-calculation/