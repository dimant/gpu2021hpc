#include <stdio.h>
#include <cstdlib>

#include "hip/hip_runtime.h"


#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__global__ void vectorAdd(const float* A, const float* B, float* C, int numElements);

inline void cudaCheckError(hipError_t error)
{
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void allocateHostVectors(float** A, float** B, float** C, int numElements)
{
    size_t size = numElements * sizeof(float);

    *A = (float*)malloc(size);
    *B = (float*)malloc(size);
    *C = (float*)malloc(size);

    if (A == NULL || B == NULL || C == NULL)
    {
        fprintf(stderr, "failed to allocate host vectors");
        exit(EXIT_FAILURE);
    }
}

void allocateDeviceVectors(float** A, float** B, float** C, int numElements)
{
    size_t size = numElements * sizeof(float);

    cudaCheckError(hipMalloc(A, size));
    cudaCheckError(hipMalloc(B, size));
    cudaCheckError(hipMalloc(C, size));
}

int main(int argc, char** argv)
{
    float* h_A, * d_A;
    float* h_B, * d_B;
    float* h_C, * d_C;

    int numElements = 50000;

    allocateHostVectors(&h_A, &h_B, &h_C, numElements);

    for (int i = 0; i < numElements; i++)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    allocateDeviceVectors(&d_A, &d_B, &d_C, numElements);

    size_t size = numElements * sizeof(float);

    cudaCheckError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd CUDA_KERNEL(blocksPerGrid, threadsPerBlock) (d_A, d_B, d_C, numElements);

    cudaCheckError(hipGetLastError());

    cudaCheckError(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    cudaCheckError(hipFree(d_A));
    cudaCheckError(hipFree(d_B));
    cudaCheckError(hipFree(d_C));

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}