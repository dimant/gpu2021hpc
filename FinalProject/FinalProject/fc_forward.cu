
#include <hip/hip_runtime.h>
#include <math.h>

#define ACT_RELU      0
#define ACT_TANH      1
#define ACT_SIGMOID   2
#define ACT_SOFTMAX   3
#define ACT_SOFTMAX10 4

__device__ __host__ __forceinline__ float relu(float f)
{
    return (fmaxf(0, f));
}

__device__ __host__ __forceinline__ float softmax10(float* fvec10, int ti)
{
    float sum = 0;
    float target = expf(fvec10[ti]);

#pragma unroll
    for (int i = 0; i < 10; i++)
    {
        sum += expf(fvec10[i]);
    }
    return (target / sum);
}

extern "C" __global__ void fc_forward(float* X, float* W, float* b, float* Z, float* Y, int n, int activation)
{
    int wi = threadIdx.x;

    Z[wi] = b[wi];

    for (int xi = 0; xi < n; xi++)
    {
        Z[wi] += X[xi] * W[wi * n + xi];
    }

    switch (activation)
    {
        case ACT_RELU:
            Y[wi] = relu(Z[wi]);
            break;
        case ACT_SOFTMAX10:
            Y[wi] = relu(Z[wi]);
            break;
    }

}
