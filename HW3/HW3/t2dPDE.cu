#include "hip/hip_runtime.h"
// index into (flattened) 1D array from 2D space
// num = number of columns per row
#define I2D(ncols, col, row) ((row)*(ncols)+(col))

extern "C" __global__
void t2dPDE_center_step(int ncols, int nrows, float alpha, float* temp_in, float* temp_out)
{
    int i00, im10, ip10, i0m1, i0p1;
    float d2tdx2, d2tdy2;

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // loop over all points in domain (except boundary)
    if (col > 0 && col < ncols - 1 && 
        row > 0 && row < nrows - 1)
    {
        // find indices into linear memory
        // for central point and neighbours
        i00 = I2D(ncols, col, row);
        ip10 = I2D(ncols, col + 1, row);
        im10 = I2D(ncols, col - 1, row);
        i0p1 = I2D(ncols, col, row + 1);
        i0m1 = I2D(ncols, col, row - 1);

        // evaluate derivatives
        d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
        d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];

        // update temperatures
        temp_out[i00] = temp_in[i00] + alpha * (d2tdx2 + d2tdy2);
    }
}

extern "C" __global__
void t2dPDE_center(int threads, int blocks,
    int nsteps, int ncols, int nrows, float alpha, float* temp_in, float* temp_out)
{
    dim3 gridSize(blocks, blocks);
    dim3 blockSize(threads, threads);

    float* left = temp_in;
    float* right = temp_out;
    float* swap = 0;

    for (int i = 0; i < nsteps; i++)
    {
        t2dPDE_center_step<<<gridSize, blockSize>>>(ncols, nrows, alpha, left, right);
        hipDeviceSynchronize();

        swap = left;
        left = right;
        right = swap;
    }
}