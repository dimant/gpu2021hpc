
#include <hip/hip_runtime.h>
// https://forums.developer.nvidia.com/t/why-am-i-getting-better-performance-with-per-column-vs-per-row-for-matrix-addition/48774
//
// This has to do with memory coalescing in CUDA, i.e. efficient use of the memory subsystem.
// When each thread is reading a column of data, then adjacent threads in a warp, at 
// each memory read instruction, are loading adjacent data from memory.This is the most 
// optimal usage of the memory subsystem.
// When each thread is reading a row of data, then adjacent threads in a warp are requesting 
// data that is separated by the row width.This is less efficient.
// This presentation may be of interest : 
// http://on-demand.gputechconf.com/gtc/2012/presentations/S0514-GTC2012-GPU-Performance-Analysis.pdf 25
// It�s necessary to think about what adjacent threads in a warp are doing instruction - by - 
// instruction, in order to understand coalescing.

extern "C" __global__ void matAdd(const float* A, const float* B, float* C, size_t nrows, size_t ncols)
{
    int stridex = blockDim.x * gridDim.x;
    int stridey = blockDim.y * gridDim.y;

    for (int row = blockDim.y * blockIdx.y + threadIdx.y; row < nrows; row += stridey)
    {
        for (int col = blockDim.x * blockIdx.x + threadIdx.x; col < ncols; col += stridex)
        {
            int idx = row * ncols + col;
            C[idx] = A[idx] + B[idx];
        }
    }
}

/// <summary>
/// row-wise matrix addition.
/// It is row-wise because every thread iterates over each column of a single row.
/// </summary>
extern "C" __global__ void matAddRow(const float* A, const float* B, float* C, size_t nrows, size_t ncols)
{
    int stridey = blockDim.y * gridDim.y;

    for (int row = blockDim.y * blockIdx.y + threadIdx.y; row < nrows; row += stridey)
    {
        for (int col = 0; col < ncols; col++)
        {
            int idx = row * ncols + col;
            C[idx] = A[idx] + B[idx];
        }
    }
}

/// <summary>
/// col-wise matrix addition.
/// It is col-wise because every thread iterates over each row of a single column.
/// </summary>
extern "C" __global__ void matAddCol(const float* A, const float* B, float* C, size_t nrows, size_t ncols)
{
    int stridex = blockDim.x * gridDim.x;

    for (int col = blockDim.x * blockIdx.x + threadIdx.x; col < ncols; col += stridex)
    {
        for (int row = 0; row < nrows; row++)
        {
            int idx = row * ncols + col;
            C[idx] = A[idx] + B[idx];
        }
    }
}

