#include "hip/hip_runtime.h"
// index into (flattened) 1D array from 2D space
// num = number of columns per row
#define I2D(ncols, col, row) ((row)*(ncols)+(col))

extern "C" __global__
void t2dPDE_center_step(int ncols, int nrows, float alpha, float* temp_in, float* temp_out)
{
    int i00, im10, ip10, i0m1, i0p1;
    float d2tdx2, d2tdy2;

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < ncols && row < nrows)
    {
        i00 = I2D(ncols, col, row);

        if (col == 0)
        {
            ip10 = I2D(ncols, col + 1, row);
            d2tdx2 = 0.0f - 2 * temp_in[i00] + temp_in[ip10];
        }
        else if (col == ncols - 1)
        {
            im10 = I2D(ncols, col - 1, row);
            d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + 0.0f;
        }
        else
        {
            ip10 = I2D(ncols, col + 1, row);
            im10 = I2D(ncols, col - 1, row);
            d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
        }

        if (row == 0)
        {
            i0p1 = I2D(ncols, col, row + 1);
            d2tdy2 = 0.0f - 2 * temp_in[i00] + temp_in[i0p1];
        }
        else if (row == nrows - 1)
        {
            i0m1 = I2D(ncols, col, row - 1);
            d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + 0.0f;
        }
        else
        {
            i0p1 = I2D(ncols, col, row + 1);
            i0m1 = I2D(ncols, col, row - 1);
            d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];
        }

        // update temperatures
        temp_out[i00] = temp_in[i00] + alpha * (d2tdx2 + d2tdy2);
    }
}

extern "C" __global__
void t2dPDE_center(int threads, int blocks,
    int nsteps, int ncols, int nrows, float alpha, float* temp_in, float* temp_out)
{
    dim3 gridSize(blocks, blocks);
    dim3 blockSize(threads, threads);

    float* left = temp_in;
    float* right = temp_out;
    float* swap = 0;

    for (int i = 0; i < nsteps; i++)
    {
        t2dPDE_center_step<<<gridSize, blockSize>>>(ncols, nrows, alpha, left, right);
        hipDeviceSynchronize();

        swap = left;
        left = right;
        right = swap;
    }
}

extern "C" __global__
void t2dPDE_full_step(int ncols, int nrows, float alpha, float* temp_in, float* temp_out)
{
    int i00, im10, ip10, i0m1, i0p1, ip20, im20, i0p2, i0m2;
    float d2tdx2, d2tdy2;

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < ncols && row < nrows)
    {
        i00 = I2D(ncols, col, row);

        if (col == 0)
        {
            ip10 = I2D(ncols, col + 1, row);
            ip20 = I2D(ncols, col + 2, row);
            d2tdx2 = temp_in[ip20] - 2 * temp_in[ip10] + temp_in[i00];
        }
        else if (col == ncols - 1)
        {
            im10 = I2D(ncols, col - 1, row);
            im20 = I2D(ncols, col - 2, row);
            d2tdx2 = temp_in[i00] - 2 * temp_in[im10] + temp_in[im20];
        }
        else
        {
            ip10 = I2D(ncols, col + 1, row);
            im10 = I2D(ncols, col - 1, row);
            d2tdx2 = temp_in[im10] - 2 * temp_in[i00] + temp_in[ip10];
        }

        if (row == 0)
        {
            i0p1 = I2D(ncols, col, row + 1);
            i0p2 = I2D(ncols, col, row + 2);
            d2tdy2 = temp_in[i0p2] - 2 * temp_in[i0p1] + temp_in[i00];
        }
        else if (row == nrows - 1)
        {
            i0m1 = I2D(ncols, col, row - 1);
            i0m2 = I2D(ncols, col, row - 2);
            d2tdy2 = temp_in[i00] - 2 * temp_in[i0m1] + temp_in[i0m2];
        }
        else
        {
            i0p1 = I2D(ncols, col, row + 1);
            i0m1 = I2D(ncols, col, row - 1);
            d2tdy2 = temp_in[i0m1] - 2 * temp_in[i00] + temp_in[i0p1];
        }

        // update temperatures
        temp_out[i00] = temp_in[i00] + alpha * (d2tdx2 + d2tdy2);
    }
}

extern "C" __global__
void t2dPDE_full(int threads, int blocks,
    int nsteps, int ncols, int nrows, float alpha, float* temp_in, float* temp_out)
{
    dim3 gridSize(blocks, blocks);
    dim3 blockSize(threads, threads);

    float* left = temp_in;
    float* right = temp_out;
    float* swap = 0;

    for (int i = 0; i < nsteps; i++)
    {
        t2dPDE_full_step << <gridSize, blockSize >> > (ncols, nrows, alpha, left, right);
        hipDeviceSynchronize();

        swap = left;
        left = right;
        right = swap;
    }
}