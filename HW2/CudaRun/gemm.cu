
#include <hip/hip_runtime.h>
// assumes square matrices (M = K = N)
// Note: A and B are source matrices
// A is M rows by K columns
// B is K rows by N columns
// C is destination
// C is M rows by N columns
extern "C" __global__ void  sgemm(
	const float* A,
	const float* B,
	float* C,
	int widthA,
	int widthB)
{
	const int col = threadIdx.x + blockIdx.x * blockDim.x;
	const int row = threadIdx.y + blockIdx.y * blockDim.y;

	if (row < widthB && col < widthA)
	{
		float sum = 0.0f;

		for (int i = 0; i < widthA; i++)
		{
			sum += 
				A[i + row * widthA] * 
				B[col + i * widthB];
		}


		C[col + row * widthB] = sum;
	}
}