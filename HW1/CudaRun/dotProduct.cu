
#include <hip/hip_runtime.h>
const int threadsPerBlock = 256;

extern "C" __global__ void dotProductFloat(const float* A, const float* B, float* C, int size)
{
	__shared__ float sumBuffer[threadsPerBlock];

	int sumBufferIdx = threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	float strideSum = 0;

	for (int cellIdx = blockIdx.x * blockDim.x + threadIdx.x; cellIdx < size; cellIdx += stride)
	{
		strideSum = strideSum + A[cellIdx] * B[cellIdx];
	}

	sumBuffer[sumBufferIdx] = strideSum;

	__syncthreads();

	for (int i = blockDim.x / 2; i != 0; i /= 2)
	{
		if (sumBufferIdx < i)
		{
			sumBuffer[sumBufferIdx] += sumBuffer[sumBufferIdx + i];
		}

		__syncthreads();
	}

	if (threadIdx.x == 0)
	{
		C[blockIdx.x] = sumBuffer[0];
	}
}

extern "C" __global__ void dotProductFloat2(const float2* A, const float2* B, float* C, int size)
{
	__shared__ float sumBuffer[threadsPerBlock];

	int sumBufferIdx = threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	float strideSum = 0;

	for (int cellIdx = blockIdx.x * blockDim.x + threadIdx.x; cellIdx < size; cellIdx += stride)
	{
		strideSum += A[cellIdx].x * B[cellIdx].x;
		strideSum += A[cellIdx].y * B[cellIdx].y;
	}

	sumBuffer[sumBufferIdx] = strideSum;

	__syncthreads();

	for (int i = blockDim.x / 2; i != 0; i /= 2)
	{
		if (sumBufferIdx < i)
		{
			sumBuffer[sumBufferIdx] += sumBuffer[sumBufferIdx + i];
		}

		__syncthreads();
	}

	if (threadIdx.x == 0)
	{
		C[blockIdx.x] = sumBuffer[0];
	}
}

extern "C" __global__ void dotProductFloat4(const float4 * A, const float4 * B, float* C, int size)
{
	__shared__ float sumBuffer[threadsPerBlock];

	int sumBufferIdx = threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	float strideSum = 0;

	for (int cellIdx = blockIdx.x * blockDim.x + threadIdx.x; cellIdx < size; cellIdx += stride)
	{
		strideSum += A[cellIdx].x * B[cellIdx].x;
		strideSum += A[cellIdx].y * B[cellIdx].y;
		strideSum += A[cellIdx].z * B[cellIdx].z;
		strideSum += A[cellIdx].w * B[cellIdx].w;
	}

	sumBuffer[sumBufferIdx] = strideSum;

	__syncthreads();

	for (int i = blockDim.x / 2; i != 0; i /= 2)
	{
		if (sumBufferIdx < i)
		{
			sumBuffer[sumBufferIdx] += sumBuffer[sumBufferIdx + i];
		}

		__syncthreads();
	}

	if (threadIdx.x == 0)
	{
		C[blockIdx.x] = sumBuffer[0];
	}
}