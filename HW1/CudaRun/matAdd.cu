
#include <hip/hip_runtime.h>
// https://forums.developer.nvidia.com/t/why-am-i-getting-better-performance-with-per-column-vs-per-row-for-matrix-addition/48774
//
// This has to do with memory coalescing in CUDA, i.e. efficient use of the memory subsystem.
// When each thread is reading a column of data, then adjacent threads in a warp, at 
// each memory read instruction, are loading adjacent data from memory.This is the most 
// optimal usage of the memory subsystem.
// When each thread is reading a row of data, then adjacent threads in a warp are requesting 
// data that is separated by the row width.This is less efficient.
// This presentation may be of interest : 
// http://on-demand.gputechconf.com/gtc/2012/presentations/S0514-GTC2012-GPU-Performance-Analysis.pdf 25
// It�s necessary to think about what adjacent threads in a warp are doing instruction - by - 
// instruction, in order to understand coalescing.

extern "C" __global__ void matAdd(const float* A, const float* B, float* C, int nrows, int ncols)
{
    int stridex = blockDim.x * gridDim.x;
    int stridey = blockDim.y * gridDim.y;

    for (int row = blockDim.y * blockIdx.y + threadIdx.y; row < nrows; row += stridey)
    {
        for (int col = blockDim.x * blockIdx.x + threadIdx.x; col < ncols; col += stridex)
        {
            int idx = row * ncols + col;
            C[idx] = A[idx] + B[idx];
        }
    }
}

extern "C" __global__ void matAddRow(const float* A, const float* B, float* C, int nrows, int ncols)
{
    int stridey = blockDim.y * gridDim.y;

    for (int row = blockDim.y * blockIdx.y + threadIdx.y; row < nrows; row += stridey)
    {
        for (int col = 0; col < ncols; col++)
        {
            int idx = row * ncols + col;
            C[idx] = A[idx] + B[idx];
        }
    }
}

extern "C" __global__ void matAddCol(const float* A, const float* B, float* C, int nrows, int ncols)
{
    int stridex = blockDim.x * gridDim.x;

    for (int col = blockDim.x * blockIdx.x + threadIdx.x; col < ncols; col += stridex)
    {
        for (int row = 0; row < nrows; row++)
        {
            int idx = row * ncols + col;
            C[idx] = A[idx] + B[idx];
        }
    }
}

